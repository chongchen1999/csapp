
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel function to print from the GPU
__global__ void hello_world() {
    printf("GPU: Hello world!\n");
}

int main(int argc, char** argv) {
    printf("CPU: Hello world!\n");
    hello_world<<<1, 10>>>();
    hipDeviceReset();
    return 0;
}
